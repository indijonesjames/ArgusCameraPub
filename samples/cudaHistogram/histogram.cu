#include "hip/hip_runtime.h"

/**
 * Based on CUB histogram code: https://github.com/NVlabs/cub/tree/master/experimental/histogram
 */

#include <stdio.h>
#include "histogram.h"

// First-pass histogram kernel (binning into privatized counters)
template <
    int         NUM_PARTS,
    int         NUM_BINS>
__global__ void histogram_smem_atomics(
    hipSurfaceObject_t surface,
    unsigned int width,
    unsigned int height,
    unsigned int *out)
{
    // global position and size
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nx = blockDim.x * gridDim.x;
    int ny = blockDim.y * gridDim.y;

    // threads in workgroup
    int t = threadIdx.x + threadIdx.y * blockDim.x; // thread index in workgroup, linear in 0..nt-1
    int nt = blockDim.x * blockDim.y; // total threads in workgroup

    // group index in 0..ngroups-1
    int g = blockIdx.x + blockIdx.y * gridDim.x;

    // initialize smem
    __shared__ unsigned int smem[NUM_BINS];
    for (int i = t; i < NUM_BINS; i += nt)
        smem[i] = 0;

    // process pixels (updates our group's partial histogram in smem)
    for (int col = x; col < width; col += nx)
    {
        for (int row = y; row < height; row += ny)
        {
            uchar1 data;
            surf2Dread(&data, surface, col, row);

            atomicAdd(&smem[((unsigned int)data.x) % NUM_BINS], 1);
        }
    }

    __syncthreads();

    // move to our workgroup's slice of output
    out += g * NUM_PARTS;

    // store local output to global
    for (int i = t; i < NUM_BINS; i += nt)
    {
        out[i] = smem[i];
    }
}

// Second pass histogram kernel (accumulation)
template <
    int         NUM_PARTS,
    int         NUM_BINS>
__global__ void histogram_smem_accum(
    const unsigned int *in,
    int n,
    unsigned int *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > NUM_BINS)
        return; // out of range

    unsigned int total = 0;
    for (int j = 0; j < n; j++)
        total += in[i + NUM_PARTS * j];

    out[i] = total;
}

template <
    int         NUM_BINS>
float run_smem_atomics(
    hipSurfaceObject_t surface,
    unsigned int width,
    unsigned int height,
    unsigned int *h_hist)
{
    enum
    {
        NUM_PARTS = 1024
    };
    hipError_t err = hipSuccess;
    dim3 block(32, 4);
    dim3 grid(16, 16);
    int total_blocks = grid.x * grid.y;

    // allocate device histogram
    unsigned int *d_hist;
    err = hipMalloc(&d_hist, NUM_BINS * sizeof(unsigned int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_hist (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    // allocate partial histogram
    unsigned int *d_part_hist;
    err = hipMalloc(&d_part_hist, total_blocks * NUM_PARTS * sizeof(unsigned int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device d_part_hist (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    dim3 block2(128);
    dim3 grid2((NUM_BINS + block.x - 1) / block.x);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&stop);
    hipEventCreate(&start);

    hipEventRecord(start, 0);

    histogram_smem_atomics<NUM_PARTS, NUM_BINS><<<grid, block>>>(
        surface,
        width,
        height,
        d_part_hist);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch histogram_smem_atomics kernel (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    histogram_smem_accum<NUM_PARTS, NUM_BINS><<<grid2, block2>>>(
        d_part_hist,
        total_blocks,
        d_hist);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch histogram_smem_accum kernel (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    float elapsed_millis;
    hipEventElapsedTime(&elapsed_millis, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    err = hipMemcpy(h_hist, d_hist, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy into h_hist (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_part_hist);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_part_hist (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_hist);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_hist (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    return elapsed_millis;
}

float histogram(hipSurfaceObject_t surface, unsigned int width, unsigned int height,
    unsigned int *histogram)
{
    return run_smem_atomics<HISTOGRAM_BINS>(surface, width, height, histogram);
}
