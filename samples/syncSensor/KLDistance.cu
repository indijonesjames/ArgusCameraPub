#include "hip/hip_runtime.h"

#ifndef _KLDISTANCE_KERNEL_H_
#define _KLDISTANCE_KERNEL_H_

#include <stdio.h>
#include "KLDistance.h"

/**
 * CUDA Kernel Device code
 *
 * Computes the KL ratio from probability ratios.
 *
 */
__global__ void
vectorKLRatio(const unsigned int *A,
              const unsigned int *B,
              float *C,
              const int numElements,
              const int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        float a = A[i];
        float b = B[i];
        a = a/(float)size;
        b = b/(float)size;
        if ( b == 0.0f) b+= .0001; // add sigma
        if ( a != 0)
            C[i] = a * log(a/b);
        else
            C[i] = 0.0f;
    }
}

float computeKLDistance(unsigned int *histOne,
                        unsigned int *histTwo,
                        const unsigned int bins,
                        const unsigned int size,
                        float *distance)
{
    hipError_t err = hipSuccess;
    int threadsPerBlock = 256;
    int blocksPerGrid =(bins + threadsPerBlock - 1) / threadsPerBlock;

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&stop);
    hipEventCreate(&start);

    hipEventRecord(start, 0);
    unsigned int *d_histOne = NULL;
    err = hipMalloc((void **)&d_histOne, bins * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device histOne (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    unsigned int *d_histTwo = NULL;
    err = hipMalloc((void **)&d_histTwo, bins * sizeof(int));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device histTwo (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the ratio bins
    float *ratio = NULL;
    err = hipMalloc((void **)&ratio, bins * sizeof(float));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector ratio (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_histOne, histOne, bins, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy into device vector histOne (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_histTwo, histTwo, bins, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy into device vector histTwo (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    vectorKLRatio<<<blocksPerGrid, threadsPerBlock>>>(d_histOne, d_histTwo, ratio, bins, size);
    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    float *dispRatio = (float *)malloc(bins * sizeof(float));
    err = hipMemcpy(dispRatio, ratio, bins * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy into device vector dispRatio (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // just sum up the vector.
    *distance = 0.0f;
    for (unsigned int i = 0; i < bins; ++i)
    {
        *distance += dispRatio[i];
    }

    err = hipFree(d_histOne);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector histOne (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_histTwo);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector histTwo (error code %s)!\n",
                         hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(ratio);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector ratio (error code %s)!\n",
                        hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    free(dispRatio);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsed_millis;
    hipEventElapsedTime(&elapsed_millis, start, stop);
    return elapsed_millis;
}
#endif
